#include "hip/hip_runtime.h"
#include <cstdio>
#include <mma.h>
#include "matmul.cuh"
#include "helpers.h"
#include "goldenSeq.h"
#include "matmul-tensor.cuh"
#include "hip/hip_fp16.h"
#include <cassert>


#define WARP_SIZE 32



template <typename elmT, typename elmAccT = elmT>
long int benchmark_tiled_tensor_mmm(
        int n_runs,
        elmT *A_device,
        elmT *B_device,
        elmAccT *ResMat_device,
        int m,
        int n,
        int k)
{
//    TODO: calculate maximum possible block_tiles_k based on shared memory size? similarly calculate warp_tiles_k based on availible registers
//    TODO: calculate m and n dimensions based on optimal block size?
//    constexpr int wmma_m = 16;
//    constexpr int wmma_n = 16;
//    constexpr int wmma_k = 16;
//
//    constexpr int warp_tiles_m = 1;
//    constexpr int warp_tiles_n = 1;
//    constexpr int warp_tiles_k = 1;
//
//    constexpr int block_tiles_m = 2;
//    constexpr int block_tiles_n = 2;
//    constexpr int block_tiles_k = 2;

// TODO: also set type using compiler options
// Set constants using compiler options
#ifdef WMMA_M
    constexpr int wmma_m = WMMA_M;
#else
    constexpr int wmma_m = 16;
#endif
#ifdef WMMA_N
    constexpr int wmma_n = WMMA_N;
#else
    constexpr int wmma_n = 16;
#endif
#ifdef WMMA_K
    constexpr int wmma_k = WMMA_K;
#else
    constexpr int wmma_k = 16;
#endif
#ifdef WARP_TILES_M
    constexpr int warp_tiles_m = WARP_TILES_M;
#else
    constexpr int warp_tiles_m = 2;
#endif
#ifdef WARP_TILES_N
    constexpr int warp_tiles_n = WARP_TILES_N;
#else
    constexpr int warp_tiles_n = 2;
#endif
#ifdef WARP_TILES_K
    constexpr int warp_tiles_k = WARP_TILES_K;
#else
    constexpr int warp_tiles_k = 2;
#endif
#ifdef BLOCK_TILES_M
    constexpr int block_tiles_m = BLOCK_TILES_M;
#else
    constexpr int block_tiles_m = 2;
#endif
#ifdef BLOCK_TILES_N
    constexpr int block_tiles_n = BLOCK_TILES_N;
#else
    constexpr int block_tiles_n = 2;
#endif
#ifdef BLOCK_TILES_K
    constexpr int block_tiles_k = BLOCK_TILES_K;
#else
    constexpr int block_tiles_k = 2;
#endif



    constexpr unsigned int threads_per_block = block_tiles_m * block_tiles_n * WARP_SIZE;
    printf("Threads used: %d\n", threads_per_block);
    assert(threads_per_block <= 1024);
    //    Assumes num_warps >= block_tiles_m * block_tiles_n, i.e. all block tiles are handled by a warp
    assert(threads_per_block / WARP_SIZE >= block_tiles_m * block_tiles_n);
//    TODO: try more than one tile per warp? would allow increasing sharing without increasing block size, but maybe this is already done by k dimension tiling?

    int dimx = ceil(((float) n)/(wmma_n * warp_tiles_n * block_tiles_n));
    int dimy = ceil(((float) m)/(wmma_m * warp_tiles_m * block_tiles_m));

    dim3 grid(dimx, dimy, 1);
    dim3 block(threads_per_block, 1, 1);

    TimeMeasurement t;

    t.start();
    for (int i = 0; i < n_runs; i++) {
        matMulTiledTensor<elmAccT, elmT, wmma_m, wmma_n, wmma_k, warp_tiles_m, warp_tiles_n, warp_tiles_k, block_tiles_m, block_tiles_n, block_tiles_k, threads_per_block><<<grid, block>>>(
                A_device, B_device, ResMat_device, m, n, k
        );
    }
    hipDeviceSynchronize();
    t.stop();

    // Check if kernel launch was successfull
    gpuAssert(hipPeekAtLastError());
    return t.elapsed();
}


template <typename elmT, int tile_size, int reg_size>
long int benchmark_tiled_mmm(
        int n_runs,
        elmT *A_device,
        elmT *B_device,
        elmT *ResMat_device,
        int m,
        int n,
        int k)
{
    int dimy = ceil( ((float) n)/(tile_size * reg_size));
    int dimx = ceil( ((float) m)/(tile_size * reg_size));
    TimeMeasurement t;
    dim3 grid(dimx, dimy, 1);
    dim3 block(16, 16, 1);

    t.start();
    for (int i = 0; i < n_runs; i++) {
        matMulTiled<elmT, tile_size, reg_size, tile_size, reg_size, tile_size><<<grid, block>>>(
                A_device, B_device, ResMat_device, m, n, k);
    }
    hipDeviceSynchronize();
    t.stop();
    // Check if kernel launch was successfull
    gpuAssert(hipPeekAtLastError());
    return t.elapsed();
}

// Expects A to have shape K x K and B to have K x N
template <typename elmT, int tile_size, int reg_size, int MatDim, bool use_tensor_cores, typename elmAccT = elmT>
//int reg_size, int n_runs = 1, int MatDim = 2, class accT = elmT>
RandomMatrix<elmAccT, MatDim>* run_mmm_kernel(
        int n_runs,
        int m,
        int n,
        int k,
        RandomMatrix<elmT, MatDim> &A,
        RandomMatrix<elmT, MatDim> &B)
{
    double total_ops = 2.0f * n * k * m;
    auto ResMat = new RandomMatrix<elmAccT, MatDim>;
    // This took me like 2 hours to fix...
    ResMat->fill(0, m, n);

    auto A_device = A.to_gpu();
    auto B_device = B.to_gpu();

    auto ResMat_device = ResMat->to_gpu();
    long int total_elapsed;
    if constexpr(use_tensor_cores) {
        total_elapsed = benchmark_tiled_tensor_mmm<elmT, elmAccT>(
                n_runs, A_device, B_device, ResMat_device, m, n, k
        );
    }
    else {
        total_elapsed = benchmark_tiled_mmm<elmT, tile_size, reg_size>(
                n_runs, A_device, B_device, ResMat_device, m, n, k
        );
    }

    hipMemcpy(ResMat->to_cpu(), ResMat_device, ResMat->flatSize() * sizeof(elmAccT), hipMemcpyDeviceToHost);
    hipFree(A_device); hipFree(B_device); hipFree(ResMat_device);
    gpuAssert(hipPeekAtLastError());


    if (!total_elapsed) {
        printf("Kernel launch failed\n");
        memset(ResMat->to_cpu(), 0, m * n);
    } else {
        printGFlops(total_elapsed, total_ops * n_runs);
    }
    return ResMat;
}





int main(int argc, char * argv[])
{
    constexpr int m = 16 * 256;
    constexpr int n = 16 * 256;
//    TODO: does not work if this is different, fix that
    constexpr int k = 16 * 256;

    int n_runs;

    if (argc == 2)
    {
        n_runs = atoi(argv[1]);
    } else {
        n_runs = 10;
    }

    // Tiled GPU verion
    // TODO: this fails when the type is float since it is not supported for wmma
    // and the templated function is still created
    RandomMatrix<float, 2> A;
    RandomMatrix<float, 2> B;
    TimeMeasurement t;
    A.fill_rand<float_range>(m, k);
    B.fill_rand<float_range>(k, n);

    std::cout << "-----" << std::endl;
    std::cout << "Running GPU register tiled version" << std::endl;
    std::cout << "Dry run" << std::endl;
    run_mmm_kernel<float, 16, 5, 2, false>(
            1, m, n, k, A, B
    );
    std::cout << "Average run of: " << n_runs << std::endl;
    RandomMatrix<float, 2> *C = run_mmm_kernel<float, 16, 5, 2, false>(
            n_runs, m, n, k, A, B
    );
    RandomMatrix<float, 2> target_res;
    target_res.fill_from(*C, m * n);
    std::cout << "-----" << std::endl;

    // GPU version
    RandomMatrix<half, 2> A_half;
    RandomMatrix<half, 2> B_half;
    A_half.fill_from(A, m, k);
    B_half.fill_from(B, k, n);

    constexpr int block_tile_size = 5; // TODO: calculate based on amount of shared memory

    std::cout << "-----" << std::endl;
    std::cout << "Running GPU tensor version" << std::endl;
    std::cout << "Dry run" << std::endl;
    RandomMatrix<float, 2> *GPU_res_tensor_half = run_mmm_kernel<half, 16, 5, 2, true, float>(
            1, m, n, k, A_half, B_half
    );

    RandomMatrix<float, 2> GPU_res_tensor;
    GPU_res_tensor.fill_from(*GPU_res_tensor_half, m, n);
    Validator<float> validator(target_res.to_cpu(), GPU_res_tensor.to_cpu(), m * n);
    // validator.setEps(0.000005); // original used by cosmin
    validator.setEps(0.0005);

    std::cout << "Average run after: " << n_runs << " runs"<< std::endl;
    run_mmm_kernel<half, 16, 5, 2, true, float>(
            n_runs, m, n, k, A_half, B_half
    );
    std::cout << "-----" << std::endl;

    validator.validate();

    delete GPU_res_tensor_half;

    return 0;



//    constexpr int k = 16 * 256;// Multiple of 8 to allign with frame leading dimension
//    constexpr int m = 16 * 256;// Multiple of 8 to allign with frame leading dimension
//    constexpr int n = 16 * 256;// Multiple of 8 to allign with frame leading dimension
//
//    // Tiled GPU verion
//    // TODO: this fails when the type is float since it is not supported for wmma
//    // and the templated function is still created
//    RandomMatrix<float, 2> A;
//    RandomMatrix<float, 2> B;
//    RandomMatrix<float, 2> CPU_res;
//    TimeMeasurement t;
//    A.fill_rand<float_range>(m, k);
//    B.fill_rand<float_range>(k, n);
//    CPU_res.fill_rand<float_range>(m, n);
//    std::cout << "Running GPU version" << std::endl;
//
//
//     RandomMatrix<float, 2> *CPU = run_mmm_kernel<float, 16, 5, 2, 1, false>(
//         m, n, k, A, B
//     );
//     RandomMatrix<float, 2> GPU_res_tiled;
//     GPU_res_tiled.fill_from(*GPU_res_tiled_half, m * n);
//
//    // GPU version
//    RandomMatrix<half, 2> A_half;
//    RandomMatrix<half, 2> B_half;
//    A_half.fill_from(A, m, k);
//    B_half.fill_from(B, k, n);
//    constexpr int block_tile_size = 5; // TODO: calculate based on amount of shared memory
//    std::cout << "Running GPU tensor version" << std::endl;
//
////    TODO: check arguments
//    RandomMatrix<half, 2> *GPU_res_tensor_half = run_mmm_kernel<half, 16, 5, 2, 1, true>(
//        m, k, n, A_half, B_half
//    );
//
//    RandomMatrix<float, 2> GPU_res_tensor;
//    GPU_res_tensor.fill_from(*GPU_res_tensor_half, m, n);
//
//    Validator<float> validator(CPU_res.to_cpu(), GPU_res_tensor.to_cpu(), m * n);
//
////    print C:
////    printf("C CPU:\n");
////    for (int i = 0; i < m; i++) {
////        for (int j = 0; j < n; j++) {
////            std::cout << CPU_res.to_cpu()[i * n + j] << " ";
////        }
////        std::cout << std::endl;
////    }
////
////    printf("C GPU:\n");
////    for (int i = 0; i < m; i++) {
////        for (int j = 0; j < n; j++) {
////            std::cout << GPU_res_tensor.to_cpu()[i * n + j] << " ";
////        }
////        std::cout << std::endl;
////    }
//
//    // validator.setEps(0.000005);
//    validator.setEps(0.05);
//    validator.validate();
//    delete GPU_res_tensor_half;
//    // delete GPU_res_tiled_half;
//
//    return 0;
}
