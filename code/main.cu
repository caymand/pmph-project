#include "hip/hip_runtime.h"
#include <cstdio>
#include <mma.h>
#include "matmul.cuh"
#include "matmul-tensor.cu"
#include "helpers.h"
#include "goldenSeq.h"
#include "matmul-tensor.cuh"

//using namespace nvcuda;


int main(int argc, char * argv[]) {
    constexpr int float_range = RAND_MAX / 10;
    constexpr int n = 16 * 5 * 24;// Multiple of 8 to allign with frame leading dimension
    constexpr int m = 16 * 5 * 24;// Multiple of 8 to allign with frame leading dimension
    constexpr int k = 16 * 5 * 24;// Multiple of 8 to allign with frame leading dimension

    constexpr int tile_size2 = 16;
    constexpr int reg_size2 = 5;

    int dimy2 = ceil( ((float) m)/(tile_size2 * reg_size2));
    int dimx2 = ceil( ((float) n)/(tile_size2 * reg_size2));

    dim3 grid2(dimx2, dimy2, 1);
    dim3 block2(16, 16, 1);


    constexpr int n_runs = 1;
    constexpr double total_ops = 2.0f * m * k * n;

    constexpr int wmma_m = 16;
    constexpr int wmma_n = 16;
    constexpr int wmma_k = 16;


//    TODO: calculate based on amount of shared memory
    constexpr int block_tile_size = 5;

    int dimy = ceil( ((float) m)/(block_tile_size * wmma_m));
    int dimx = ceil( ((float) n)/(block_tile_size * wmma_n));

    dim3 grid(dimx, dimy, 1);
    dim3 block(16, 16, 1);
    
    // Allocate 3 matrices with random data
    RandomMatrix<half, 2> Ahost;
    RandomMatrix<half, 2> Bhost;
    RandomMatrix<half, 2> Chost;
    RandomMatrix<half, 2> Dhost;
    
    Ahost.fill<float_range>(m, k);
    Bhost.fill<float_range>(k, n);
    Chost.fill<float_range>(m, n);
    Dhost.fill<float_range>(m, n);

    TimeMeasurement t;

//    std::cout << "Running on CPU" << std::endl;
//    t.start();
//    goldenSeq<float>(Ahost.to_cpu(), Bhost.to_cpu(), Chost.to_cpu(), n, k, m);
//    t.stop();

//    printGFlops(t.elapsed(), total_ops);

    std::cout << "Running on GPU:" << std::endl;
    auto Adevice = Ahost.to_gpu();
    auto Bdevice = Bhost.to_gpu();
    auto Cdevice = Chost.to_gpu();
    auto Ddevice = Dhost.to_gpu();

    t.start();
    {
        for (int i = 0; i < n_runs; i++) {
            matMulTiled<half, tile_size2, reg_size2, tile_size2, reg_size2, tile_size2><<<grid, block>>>(
                    Adevice, Bdevice, Cdevice, m, n, k);
        }
        hipDeviceSynchronize();
    }
    t.stop();
    hipMemcpy(Chost.to_cpu(), Cdevice, Chost.flatSize() * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree(Adevice); hipFree(Bdevice); hipFree(Ddevice);
    gpuAssert( hipPeekAtLastError() );

    printGFlops(t.elapsed(), total_ops * n_runs);
    
    t.start();
    {
        for (int i = 0; i < n_runs; i++) {
            matMulTiledTensor<half, half, wmma_m, wmma_n, wmma_k, block_tile_size><<<grid, block>>>(
                Adevice, Bdevice, Ddevice, m, n, k);
        }
        hipDeviceSynchronize();
    }
    t.stop();
    hipMemcpy(Dhost.to_cpu(), Ddevice, Dhost.flatSize() * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree(Adevice); hipFree(Bdevice); hipFree(Ddevice);
    gpuAssert( hipPeekAtLastError() );

    printGFlops(t.elapsed(), total_ops * n_runs);

    Validator<float> validator(reinterpret_cast<float *>(Chost.to_cpu()), reinterpret_cast<float *>(Dhost.to_cpu()), n * k);
    validator.setEps(0.000005);
    validator.validate();

    return 0;
}
