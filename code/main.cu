#include "hip/hip_runtime.h"
#include <cstdio>
#include <mma.h>
#include "matmul.cuh"
#include "helpers.h"
#include "goldenSeq.h"
#include "matmul-tensor.cuh"
#include "hip/hip_fp16.h"
#include <cassert>
//#include <hipblas.h>
#include <hipblas.h>


#define WARP_SIZE 32
#define SHARED_MEM_SIZE 49152
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_REGISTERS_PER_BLOCK 65536

#define SHARED_PADDING 8


template <typename elmT, typename elmAccT = elmT>
long int benchmark_tiled_tensor_mmm(
        int n_runs,
        elmT *A_device,
        elmT *B_device,
        elmAccT *C_device,
        int m,
        int n,
        int k)
{
//    TODO: calculate maximum possible block_tiles_k based on shared memory size? similarly calculate warp_tiles_k based on availible registers
//    TODO: calculate m and n dimensions based on optimal block size?
//    constexpr int wmma_m = 16;
//    constexpr int wmma_n = 16;
//    constexpr int wmma_k = 16;
//
//    constexpr int warp_tiles_m = 1;
//    constexpr int warp_tiles_n = 1;
//    constexpr int warp_tiles_k = 1;
//
//    constexpr int block_tiles_m = 2;
//    constexpr int block_tiles_n = 2;
//    constexpr int block_tiles_k = 2;

// TODO: also set type using compiler options
// Set constants using compiler options
#ifdef WMMA_M
    constexpr int wmma_m = WMMA_M;
#else
    constexpr int wmma_m = 16;
#endif
#ifdef WMMA_N
    constexpr int wmma_n = WMMA_N;
#else
    constexpr int wmma_n = 16;
#endif
#ifdef WMMA_K
    constexpr int wmma_k = WMMA_K;
#else
    constexpr int wmma_k = 16;
#endif
#ifdef WARP_TILES_M
    constexpr int warp_tiles_m = WARP_TILES_M;
#else
    constexpr int warp_tiles_m = 2;
#endif
#ifdef WARP_TILES_N
    constexpr int warp_tiles_n = WARP_TILES_N;
#else
    constexpr int warp_tiles_n = 2;
#endif
#ifdef WARP_TILES_K
    constexpr int warp_tiles_k = WARP_TILES_K;
#else
    constexpr int warp_tiles_k = 2;
#endif
#ifdef BLOCK_TILES_M
    constexpr int block_tiles_m = BLOCK_TILES_M;
#else
    constexpr int block_tiles_m = 2;
#endif
#ifdef BLOCK_TILES_N
    constexpr int block_tiles_n = BLOCK_TILES_N;
#else
    constexpr int block_tiles_n = 2;
#endif
#ifdef BLOCK_TILES_K
    constexpr int block_tiles_k = BLOCK_TILES_K;
#else
    constexpr int block_tiles_k = 2;
#endif

    constexpr unsigned int threads_per_block = block_tiles_m * block_tiles_n * WARP_SIZE;
    printf("Threads used: %d/%d\n", threads_per_block, MAX_THREADS_PER_BLOCK);
    assert(threads_per_block <= MAX_THREADS_PER_BLOCK);
    //    Assumes num_warps >= block_tiles_m * block_tiles_n, i.e. all block tiles are handled by a warp
    assert(threads_per_block / WARP_SIZE >= block_tiles_m * block_tiles_n);
//    TODO: try more than one tile per warp?
//    Would allow increasing sharing without increasing block size, but maybe this is already done by k dimension tiling?
//    Would maybe allow more concurrent blocks? Would mean less threads for copying memory

    int dimx = ceil(((float) n)/(wmma_n * warp_tiles_n * block_tiles_n));
    int dimy = ceil(((float) m)/(wmma_m * warp_tiles_m * block_tiles_m));

    dim3 grid(dimx, dimy, 1);
    dim3 block(threads_per_block, 1, 1);

    printf("Blocks used: %d x %d = %d\n", dimx, dimy, dimx * dimy);

    //  TODO: calculate register usage?
    printf("Available registers per thread: %d (%d per block)\n", MAX_REGISTERS_PER_BLOCK / threads_per_block, MAX_REGISTERS_PER_BLOCK);

    constexpr unsigned int shared_m = wmma_m * warp_tiles_m * block_tiles_m;
    constexpr unsigned int shared_n = wmma_n * warp_tiles_n * block_tiles_n;
    constexpr unsigned int shared_k = wmma_k * warp_tiles_k * block_tiles_k;

//    assert(m % wmma_m == 0 && m % wmma_m * warp_tiles_m == 0 && m % wmma_m * warp_tiles_m * block_tiles_m == 0);
//    assert(n % wmma_n == 0 && n % wmma_n * warp_tiles_n == 0 && n % wmma_n * warp_tiles_n * block_tiles_n == 0);
//    assert(k % wmma_k == 0 && k % wmma_k * warp_tiles_k == 0 && k % wmma_k * warp_tiles_k * block_tiles_k == 0);
//    printf("%d %d %d\n", m % wmma_m, m % wmma_m * warp_tiles_m, m % wmma_m * warp_tiles_m * block_tiles_m);
//    printf("%d %d %d\n", n % wmma_n, n % wmma_n * warp_tiles_n, n % wmma_n * warp_tiles_n * block_tiles_n);
//    printf("%d %d %d\n", k % wmma_k, k % wmma_k * warp_tiles_k, k % wmma_k * warp_tiles_k * block_tiles_k);


    constexpr unsigned int shared_memory_used_AB = shared_m * (shared_k + SHARED_PADDING) * sizeof(elmT) + shared_k * (shared_n + SHARED_PADDING) * sizeof(elmT);
#ifdef CACHE_C
//    Add space for caching C
    constexpr unsigned int shared_memory_used = shared_memory_used_AB + shared_m * (shared_n + SHARED_PADDING) * sizeof(elmAccT);
#else
    constexpr unsigned int shared_memory_used = shared_memory_used_AB;
#endif
    printf("Shared memory used: %d/%d bytes (%.0f%%)\n", shared_memory_used, SHARED_MEM_SIZE, (float) shared_memory_used / SHARED_MEM_SIZE * 100);


    TimeMeasurement t;

    t.start();
    for (int i = 0; i < n_runs; i++) {
        matMulTiledTensor<elmT, elmAccT, wmma_m, wmma_n, wmma_k, warp_tiles_m, warp_tiles_n, warp_tiles_k, block_tiles_m, block_tiles_n, block_tiles_k, threads_per_block><<<grid, block>>>(
                A_device, B_device, C_device, m, n, k
        );
    }
    hipDeviceSynchronize();
    t.stop();

    // Check if kernel launch was successfull
    gpuAssert(hipPeekAtLastError());
    return t.elapsed();
}


template <typename elmT, typename elmAccT>
long int benchmark_tiled_mmm(
        int n_runs,
        elmT *A_device,
        elmT *B_device,
        elmAccT *C_device,
        int m,
        int n,
        int k)
{
    constexpr int tile_size = 16;
    constexpr int reg_size = 5;

    int dimy = ceil( ((float) n)/(tile_size * reg_size));
    int dimx = ceil( ((float) m)/(tile_size * reg_size));
    TimeMeasurement t;
    dim3 grid(dimx, dimy, 1);
    dim3 block(16, 16, 1);

//    hipblasHandle_t handle;
//    hipblasStatus_t stat;
//    stat = hipblasCreate(&handle);
//    half alpha = (half) 1.0;
//    half beta = (half) 0.0;
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf ("CUBLAS initialization failed\n");
//        return EXIT_FAILURE;
//    }

    t.start();
    for (int i = 0; i < n_runs; i++) {
//        hipblasHgemm(
//                handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
//                &alpha,
//                (const half *) A_device, k,
//                (const half *) B_device, n,
//                &beta,
//                (half *) C_device, n);
        matMulTiled<elmT, elmAccT, tile_size, reg_size, tile_size, reg_size, tile_size><<<grid, block>>>(
                A_device, B_device, C_device, m, n, k);
    }
    hipDeviceSynchronize();
    t.stop();
    // Check if kernel launch was successfull
    gpuAssert(hipPeekAtLastError());
    return t.elapsed();
}

// Expects A to have shape K x K and B to have K x N
template <typename elmT, typename elmAccT, int MatDim, bool use_tensor_cores>
//int reg_size, int n_runs = 1, int MatDim = 2, class accT = elmT>
void run_mmm_kernel(
        int n_runs,
        int m,
        int n,
        int k,
        RandomMatrix<elmT, MatDim> &A,
        RandomMatrix<elmT, MatDim> &B,
        RandomMatrix<elmAccT, MatDim> &C)
{
    double total_ops = 2.0f * n * k * m;

    auto A_device = A.to_gpu();
    auto B_device = B.to_gpu();

    auto C_device = C.to_gpu();
    long int total_elapsed;
    if constexpr(use_tensor_cores) {
        total_elapsed = benchmark_tiled_tensor_mmm<elmT, elmAccT>(
                n_runs, A_device, B_device, C_device, m, n, k
        );
    }
    else {
        total_elapsed = benchmark_tiled_mmm<elmT, elmAccT>(
                n_runs, A_device, B_device, C_device, m, n, k
        );
    }

    hipMemcpy(C.to_cpu(), C_device, C.flatSize() * sizeof(elmAccT), hipMemcpyDeviceToHost);
    hipFree(A_device); hipFree(B_device); hipFree(C_device);
    gpuAssert(hipPeekAtLastError());


    if (!total_elapsed) {
        printf("Kernel launch failed\n");
        memset(C.to_cpu(), 0, m * n);
    } else {
        printGFlops(total_elapsed, total_ops * n_runs);
    }
}


#ifdef ELM_T
typedef ELM_T elmT;
#else
typedef half elmT;
#endif

#ifdef ACC_T
typedef ACC_T accT;
#else
typedef float accT;
#endif


int main(int argc, char * argv[])
{
    int m = 16 * 256;
    int n = 16 * 256;
//    TODO: does not work if this is different, fix that
    int k = 16 * 256;

    int n_runs = 10;

    if (argc >= 2)
    {
        n_runs = atoi(argv[1]);
    }
    if (argc == 3)
    {
        int input_int = atoi(argv[2]);
        m = input_int;
        n = input_int;
        k = input_int;
    } else if (argc == 4)
    {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
    } else if (argc == 5)
    {
        n_runs = atoi(argv[1]);
        m = atoi(argv[2]);
        n = atoi(argv[3]);
        k = atoi(argv[4]);
    }


    TimeMeasurement t;

    //  Define matrices
    RandomMatrix<elmT, 2> A;
    RandomMatrix<elmT, 2> B;
    RandomMatrix<accT, 2> A_accT;
    RandomMatrix<accT, 2> B_accT;
    RandomMatrix<accT, 2> C;
    RandomMatrix<accT, 2> C_target;
    RandomMatrix<accT, 2> C_actual;

    //  Initialize matrices
    A.fill_rand<float_range>(m, k);
    B.fill_rand<float_range>(k, n);
    C.fill(0, m, n);
    C_target.fill(0, m, n);
    A_accT.fill_from(A, m, k);
    B_accT.fill_from(B, k, n);

    // Tiled GPU verion
    std::cout << "-----" << std::endl;
    std::cout << "Running GPU register tiled version" << std::endl;
    std::cout << "Dry run" << std::endl;
    run_mmm_kernel<accT, accT, 2, false>(
            1, m, n, k, A_accT, B_accT, C_target
    );
    std::cout << "Average run of: " << n_runs << std::endl;
    run_mmm_kernel<accT , accT, 2, false>(
            n_runs, m, n, k, A_accT, B_accT, C_target
    );

    std::cout << "-----" << std::endl;

    // GPU version
    std::cout << "-----" << std::endl;
    std::cout << "Running GPU tensor version" << std::endl;
    std::cout << "Dry run" << std::endl;
    run_mmm_kernel<elmT, accT, 2, true>(
            1, m, n, k, A, B, C
    );

    C_actual.fill_from(C, m, n);

    std::cout << "Average run after: " << n_runs << " runs"<< std::endl;
    run_mmm_kernel<elmT, accT, 2, true>(
            n_runs, m, n, k, A, B, C
    );
    std::cout << "-----" << std::endl;

    Validator<accT> validator(C_target.to_cpu(), C_actual.to_cpu(), m * n);
//    validator.setEps(0.000005); // original used by cosmin
    validator.setEps(0.0005);
//    Check if something is wrong, or we really need this eps
//    validator.setEps((accT) 0.1);

    validator.validate();

    hipFree(A.to_gpu());
    hipFree(B.to_gpu());
    hipFree(C.to_gpu());
    hipFree(C_target.to_gpu());
    hipFree(C_actual.to_gpu());
    hipFree(A_accT.to_gpu());
    hipFree(B_accT.to_gpu());

    return 0;
}

// TODO: try half-half, half-float, (double-double, tf32-tf32)
// Find best parameters for tiled and tensor for each case
// TODO: graphs

